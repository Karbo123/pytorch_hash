/*
    fnv-1a hashing for: Byte, Char, Double, Float, Int, Long, Short, Half, ComplexFloat, ComplexDouble
      input data must have shape == (num, dim)
      output data has shape == (num, )
*/
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <pybind11/pybind11.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>

template <typename scalar_t>
struct hash_functor {
    scalar_t* ptr_in;
    int64_t* ptr_out;
    uint64_t num, dim;
    hash_functor(scalar_t* ptr_in, int64_t* ptr_out, uint64_t num, uint64_t dim): 
                 ptr_in(ptr_in), ptr_out(ptr_out), num(num), dim(dim) {}
    __host__ __device__ void operator()(uint64_t index_row) {
        auto ptr_in_i = reinterpret_cast<u_char*>(ptr_in + index_row * dim);
        uint64_t number = dim * sizeof(scalar_t);
        uint64_t result = 0XCBF29CE484222325;
        while (number--) result = (result ^ (*ptr_in_i++)) * 0x00000100000001B3;
        ptr_out[index_row] = reinterpret_cast<int64_t&>(result);
    }
};

torch::Tensor hash(torch::Tensor data) {
    TORCH_CHECK(data.is_contiguous(), "data must be contiguous");
    auto num = data.size(0);
    auto dim = data.size(1);
    auto is_cuda = data.is_cuda();

    auto options = torch::TensorOptions().dtype(torch::kInt64);
    torch::Tensor out = data.new_empty({num}, options);

    AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, data.type(), "hash", [&] {
        auto functor = hash_functor<scalar_t>(data.data_ptr<scalar_t>(), 
                                              out .data_ptr<int64_t>(),
                                              num, dim);
        auto iter_rows = thrust::counting_iterator<uint64_t>(0);
        if (is_cuda) {
            auto stream = at::cuda::getCurrentCUDAStream();
            auto device = thrust::cuda::par.on(stream);
            thrust::for_each_n(device, iter_rows, num, functor);
            hipStreamSynchronize(stream);
        }
        else {
            thrust::for_each_n(thrust::host, iter_rows, num, functor);
        }
    });
    
    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("hash", &hash, "fnv-1a hashing for pytorch", pybind11::arg("data"));
}